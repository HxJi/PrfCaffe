#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/relu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReLUForward(const int n, const Dtype* in, Dtype* out,
    Dtype negative_slope, int* zero_element) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope;
    if(out[index] == 0) zero_element[index/CAFFE_CUDA_NUM_THREADS] += 1;
  }
}

template <typename Dtype>
void ReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype negative_slope = this->layer_param_.relu_param().negative_slope();

  //[houxiang]
  std::string filename = ("/home/hj14/caffe/hj_test/relu_sparsity.txt");
  std::ofstream sparsity_output;
  sparsity_output.open(filename.c_str(), ios::app);
  //count the zero number in each block to save space
  sparsity_output << count << " ";
  int block_num = CAFFE_GET_BLOCKS(count);
  int zero_cell[block_num];
  for(int i=0; i<block_num; ++i){
	  zero_cell[i] = 0;
  }
  hipError_t err = hipSuccess;
  int *dev_zero_cell;
  err = hipMalloc((void**)&dev_zero_cell, block_num * sizeof(int));
  if(err!=hipSuccess) {
        printf("the hipMalloc on GPU is failed");
   }
  hipMemcpy(dev_zero_cell, zero_cell, block_num * sizeof(int), hipMemcpyHostToDevice);

  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, negative_slope, dev_zero_cell );
  
  //[houxiang]
  hipMemcpy(&zero_cell, dev_zero_cell, block_num * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_zero_cell);
  int total_zero = 0;
  for(int i=0; i<block_num; ++i){
	      total_zero = zero_cell[i] + total_zero;
        //sparsity_output << "[" <<i<<"]:"<< zero_cell[i]<<" ";
  }
  sparsity_output << total_zero << std::endl;

  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void ReLUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * negative_slope);
  }
}

template <typename Dtype>
void ReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
    // NOLINT_NEXT_LINE(whitespace/operators)
    ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, negative_slope);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ReLULayer);


}  // namespace caffe
